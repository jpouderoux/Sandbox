#include "hip/hip_runtime.h"
#include <iostream>

#include "foo.h"

Foo::Foo()
  {
  int devid=0, num=0;
  hipGetDeviceCount(&num);
  hipGetDevice(&devid);

  std::cout << "device count" << num << std::endl;
  std::cout << "device id" << devid << std::endl;
  }
